
#include <hip/hip_runtime.h>
#include <iostream>
#include <cudnn.h>
#include <hip/hip_fp16.h>

#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

int main(int argc, char const *argv[]) {
  cudnnHandle_t cudnn;
  checkCUDNN(cudnnCreate(&cudnn));

  int N = 1, C = 32, Dx = 320, Hx = 320, Wx = 320;
  int Dy = 160, Hy = 160, Wy = 160;
  int c = 32, k = 32, t = 2, r = 2, s = 2;
  int dx_dims[] = {N, C, Dx, Hx, Wx};
  int dy_dims[] = {N, C, Dy, Hy, Wy};
  int w_dims[] = {k, c, t, r, s};
  int paddings[] = {0, 0, 0};
  int strides[] = {2, 2, 2};
  int dilations[] = {1, 1, 1};

  /* Below is to do BackwardData */
  cudnnTensorDescriptor_t dy_desc;
  checkCUDNN(cudnnCreateTensorDescriptor(&dy_desc));
  checkCUDNN(cudnnSetTensorNdDescriptorEx(/*tensorDesc=*/dy_desc,
                                          /*format=*/CUDNN_TENSOR_NHWC,
                                          /*dataType=*/CUDNN_DATA_HALF,
                                          /*nbDims=*/5,
                                          /*dimsA*/dy_dims));

  cudnnTensorDescriptor_t dx_desc;
  checkCUDNN(cudnnCreateTensorDescriptor(&dx_desc));
  checkCUDNN(cudnnSetTensorNdDescriptorEx(/*tensorDesc=*/dx_desc,
                                          /*format=*/CUDNN_TENSOR_NHWC,
                                          /*dataType=*/CUDNN_DATA_HALF,
                                          /*nbDims=*/5,
                                          /*dimsA*/dx_dims));

  cudnnFilterDescriptor_t w_desc;
  checkCUDNN(cudnnCreateFilterDescriptor(&w_desc));
  checkCUDNN(cudnnSetFilterNdDescriptor(/*filterDesc=*/w_desc,
                                        /*dataType=*/CUDNN_DATA_HALF,
                                        /*format=*/CUDNN_TENSOR_NHWC,
                                        /*nbDims=*/5,
                                        /*dimsA*/w_dims));

  cudnnConvolutionDescriptor_t conv_desc;
  checkCUDNN(cudnnCreateConvolutionDescriptor(&conv_desc));
  checkCUDNN(cudnnSetConvolutionNdDescriptor(
      /*conv_desc=*/conv_desc,
      /*arrayLength=*/3,
      /*padA=*/paddings,
      /*filterStrideA=*/strides,
      /*dilationA=*/dilations,
      /*mode=*/CUDNN_CROSS_CORRELATION,
      /*dataType=*/CUDNN_DATA_FLOAT));

  // checkCUDNN(cudnnSetConvolutionMathType(conv_desc, CUDNN_FMA_MATH));
  checkCUDNN(cudnnSetConvolutionMathType(conv_desc, CUDNN_TENSOR_OP_MATH));

  int algos[5] = {1, 2, 3, 4, 0};
  for(auto algo : algos) {
    size_t size_in_bytes = 0;
    cudnnStatus_t status = cudnnGetConvolutionBackwardDataWorkspaceSize(
        /*handle=*/cudnn,
        /*wDesc=*/w_desc,
        /*dyDesc=*/dy_desc,
        /*convDesc=*/conv_desc,
        /*dxDesc=*/dx_desc,
        /*algo=*/(cudnnConvolutionBwdDataAlgo_t)algo,
        /*sizeInBytes=*/&size_in_bytes);
    if (status != CUDNN_STATUS_SUCCESS) {
      printf("XXX algo %d failed\n", algo);
    } else {
      printf("XXX found algo %d workspace size in bytes: %ld\n", algo, size_in_bytes);
    }
  }
}
